#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>


#define FP float
#define TW 32
#define N 8


__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  __shared__ FP atile[TW][TW], 
             btile[N][TW][TW]; 
  int tx = threadIdx.x; int ty = threadIdx.y; 
  FP cvalue[N];
  int col[N];
  for(int i = 0; i < N; i++){
    col[i] = tx + blockDim.x * (blockIdx.x * N + i);
    cvalue[i] = 0;
  }
  int row = ty + blockDim.y * blockIdx.y;
  int num_tile = N; //(m - blockDim.x * blockIdx.x * N) / TW  < N? (m - blockDim.x * blockIdx.x * N) / TW: N ; 
    for(int k = 0; k < p / TW; k++){
      atile[ty][tx] = a[row * p + k * TW + tx];
      for(int i = 0; i < num_tile; i++)
        btile[i][ty][tx] = b[(k * TW + ty) * m + col[i]];
         
      __syncthreads();
      for(int i = 0; i < num_tile; i++)
        for(int l = 0; l < TW; l++) cvalue[i] += atile[ty][l] * btile[i][l][tx];

      __syncthreads();
    }
    for(int i = 0; i < num_tile; i++) 
      c[row*m + col[i]] = cvalue[i];
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {
  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1, Grid_Dim_y = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, p, m; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<5) || (argc>6)) {
    printf("Usage: matmul <matrix dim n> <matrix dim p> <matrix dim m> <block dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);
   
  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
//    exit (-1);
  }
  Grid_Dim_x = ((m + Block_Dim - 1) / Block_Dim  + N - 1) / N ; // Square grid
  Grid_Dim_y = (n + Block_Dim - 1) / Block_Dim;

  if (argc==6) {
    gpunum = atoi(argv[5]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = (%d, %d, %d)\n",n, p, m);
  printf("Block_Dim = %d, Grid_Dim_x = %d, Grid_Dim_y = %d\n",Block_Dim, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(FP); // number of bytes in total in arrays
  size_b = p * m * sizeof(FP);
  size_c = n * m * sizeof(FP);
  a = (FP*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size_b);
  c = (FP*) malloc(size_c); // results from GPU

  srand(12345);
  // int p = n; //Used here only to illustrate proper initialization for non-square case
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size_a); // allocate memory on device
  hipMalloc((void**)&dev_b, size_b);
  hipMalloc((void**)&dev_c, size_c);

  hipMemcpy(dev_a, a , size_a ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size_b ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n, p, m);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size_c ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


 cpu_matrixmult(a,b,c, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
