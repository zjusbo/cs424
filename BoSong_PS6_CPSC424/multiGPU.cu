#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>


#define FP float
#define TW 32
#define N 4
#define DEVCOUNT 2

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {

  __shared__ FP atile[TW][TW], 
             btile[N][TW][TW]; 
  int tx = threadIdx.x; int ty = threadIdx.y; 
  FP cvalue[N];
  int col[N];
  for(int i = 0; i < N; i++){
    col[i] = tx + blockDim.x * (blockIdx.x * N + i);
    cvalue[i] = 0;
  }
  int row = ty + blockDim.y * blockIdx.y;
  int num_tile = N; //(m - blockDim.x * blockIdx.x * N) / TW  < N? (m - blockDim.x * blockIdx.x * N) / TW: N ; 
  for(int k = 0; k < p / TW; k++){
    atile[ty][tx] = a[row * p + k * TW + tx];
    for(int i = 0; i < num_tile; i++)
      btile[i][ty][tx] = b[(k * TW + ty) * m + col[i]];
         
      __syncthreads();
       for(int i = 0; i < num_tile; i++)
        for(int l = 0; l < TW; l++) cvalue[i] += atile[ty][l] * btile[i][l][tx];

      __syncthreads();
  }
    for(int i = 0; i < num_tile; i++) 
      c[row*m + col[i]] = cvalue[i];
}

void cpu_matrixmult(FP *a,FP *b, FP *c, int n, int p, int m) {
  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < m; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * m + col;
      cvalue = 0.;
      for (indexa = row*p; indexa < (row*p + p); indexa++, indexb+=m) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}

int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1, Grid_Dim_y = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n, p, m; // matrix dimension
  FP *a,*b,*c_part[DEVCOUNT], *c;
  FP *dev_a[DEVCOUNT], *dev_b[DEVCOUNT], *dev_c[DEVCOUNT];
  int size_a, size_b, size_c; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<5) || (argc>6)) {
    printf("Usage: matmul <matrix dim n> <matrix dim p> <matrix dim m> <block dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);
   
  Block_Dim = atoi(argv[4]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
//    exit (-1);
  }
  Grid_Dim_x = (m / Block_Dim  + N - 1) / N; 
  Grid_Dim_y = (n / DEVCOUNT ) / Block_Dim; // divide gride by 2 vertically

  if (argc==6) {
    gpunum = atoi(argv[5]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = (%d, %d, %d)\n",n, p, m);
  printf("Block_Dim = %d, Grid_Dim_x = %d, Grid_Dim_y = %d\n",Block_Dim, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size_a = n * p * sizeof(FP); // number of bytes in total in arrays
  size_b = p * m * sizeof(FP);
  size_c = n * m * sizeof(FP);
  a = (FP*) malloc(size_a); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size_b);
  c = (FP*) malloc(size_c);
  for(int i = 0; i < DEVCOUNT; i++){
    c_part[i] = (FP*) malloc(size_c / DEVCOUNT);  
  }
  srand(12345);
  // int p = n; //Used here only to illustrate proper initialization for non-square case
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < m;j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------
  // allocate space
  for(int i = 0; i < DEVCOUNT; i++){
    hipSetDevice(i);
    hipMalloc((void**)&dev_a[i], size_a / DEVCOUNT); // allocate memory on device
    hipMalloc((void**)&dev_b[i], size_b);
    hipMalloc((void**)&dev_c[i], size_c / DEVCOUNT);
   
    hipMemcpyAsync(dev_a[i], a + i * n * p / DEVCOUNT, size_a / DEVCOUNT ,hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_b[i], b , size_b ,hipMemcpyHostToDevice);

  }

  // run kernel
  hipSetDevice(0);
  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  hipEventRecord(start, 0); 
  for(int i = 0; i < DEVCOUNT; i++){
    hipSetDevice(i);
    gpu_matrixmult<<<Grid,Block>>>(dev_a[i],dev_b[i],dev_c[i],n / DEVCOUNT, p, m); // compute the left part of matrix
  } 

  // wait for completion
  hipSetDevice(0);
  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  // copy result back to host
  for(int i = 0; i < DEVCOUNT; i++){
    hipSetDevice(i);
    hipMemcpy(c_part[i],dev_c[i], size_c / DEVCOUNT,hipMemcpyDeviceToHost);
  }

  // merge results
  for(int i = 0; i < DEVCOUNT; i++){
    memcpy(c + m * n * i / DEVCOUNT, c_part[i], size_c / DEVCOUNT);
  }
  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
  hipSetDevice(0);
  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed

  cpu_matrixmult(a,b,c, n, p, m); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*m;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  for(int i = 0; i < DEVCOUNT; i++){
    free(c_part[i]);
    hipFree(dev_a[i]);
    hipFree(dev_b[i]);
    hipFree(dev_c[i]);
  }
    hipEventDestroy(start);
    hipEventDestroy(stop);


  return 0;
}
