#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void gpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * n + col;
  
  if(col < n && row < n) {
    c[index] = 0.;
    for (int indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
      c[index] += a[indexa]*b[indexb];
  }

}


void cpu_matrixmult(FP *a,FP *b, FP *c, int n) {

  int index, indexa, indexb;
  FP cvalue;
  for(int col=0;col < n; col++)
    for(int row=0;row < n; row++) {
      indexb = col;
      index = row * n + col;
      cvalue = 0.;
      for (indexa = row*n; indexa < (row*n + n); indexa++, indexb+=n) 
	cvalue += a[indexa]*b[indexb];
      c[index] -= cvalue; //NOTE: This calculates the diff between CPU and GPU computations.
    }
}


int main(int argc, char *argv[]) {

  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim = 1; //Grid dimension, x and y, square
  int Block_Dim = 1; //Block dimension, x and y, square

  int n; // matrix dimension
  FP *a,*b,*c;
  FP *dev_a, *dev_b, *dev_c;
  int size; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // --------------------SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
     printf("Device count = %d\n",gpucount);
  }

  if ((argc<4) || (argc>5)) {
    printf("Usage: matmul <matrix dim> <block dim> <grid dim> [<dev num>]\n");
    exit (-1);
  }

  n = atoi(argv[1]);

  Block_Dim = atoi(argv[2]); // Square block
  if (Block_Dim*Block_Dim > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim = atoi(argv[3]); // Square grid
  if (Grid_Dim*Block_Dim < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit (-1);
  }

  if (argc==5) {
    gpunum = atoi(argv[4]); // Device number
    if ((gpunum > 2) || (gpunum < 0)) {
      printf("Error, Device number must be 0, 1, or 2\n");
      exit (-1);
    }
  }
  hipSetDevice(gpunum);
  printf("Using device %d\n",gpunum);
  
  printf("Matrix Dimension = %d\n",n);
  printf("Block_Dim = %d, Grid_Dim = %d\n",Block_Dim,Grid_Dim);

  dim3 Grid(Grid_Dim, Grid_Dim); //Grid structure
  dim3 Block(Block_Dim, Block_Dim); //Block structure

  size = n * n * sizeof(FP); // number of bytes in total in arrays

  a = (FP*) malloc(size); // dynamically allocated memory for arrays on host
  b = (FP*) malloc(size);
  c = (FP*) malloc(size); // results from GPU

  srand(12345);
  int p = n; //Used here only to illustrate proper initialization for non-square case
  for(i=0;i < n;i++)
    for(j=0;j < p;j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
      //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
    }

  for(i=0;i < p;i++)
    for(j=0;j < n;j++) {
      b[i * n + j] = (FP) rand() / (FP) RAND_MAX;
      //      b[i * n + j] = (FP) i+j; // may be helpful for debugging
    }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**)&dev_a, size); // allocate memory on device
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);

  hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,n);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  hipMemcpy(c,dev_c, size ,hipMemcpyDeviceToHost);

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
  // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

  hipEventRecord(start, 0); // use same timing
  // cudaEventSynchronize(start); // not needed


  cpu_matrixmult(a,b,c, n); // do calculation on host (NOTE: This computes the diff with GPU result.)

  hipEventRecord(stop, 0); // instrument code to measue end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop );

  printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

// ------------------- check device creates correct results -----------------

  double error, suma, sumb, sumc, ai, bi, ci;
  suma = 0.; sumb = 0; sumc = 0;
  for(i=0;i < n*n;i++) {
    ai = (double) a[i];
    bi = (double) b[i];
    ci = (double) c[i];
    suma += ai*ai;
    sumb += bi*bi;
    sumc += ci*ci;
  }
  suma = sqrt(suma);
  sumb = sqrt(sumb);
  sumc = sqrt(sumc);
  error =  sumc/(n*suma*sumb);
  printf("Scaled error between GPU and CPU: %e\n", error);

// -------------- clean up ---------------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
